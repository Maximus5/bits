/*****************************************************************************
bits_enumerate_cuda.cu
(c) 2012 - Ryan M. Layer
Hall Laboratory
Quinlan Laboratory
Department of Computer Science
Department of Biochemistry and Molecular Genetics
Department of Public Health Sciences and Center for Public Health Genomics,
University of Virginia
rl6sf@virginia.edu

Licenced under the GNU General Public License 2.0 license.
******************************************************************************/
#include "lineFileUtilities.h"
#include "bits_enumerate_cuda.h"
#include "interval_cuda.h"
#include "timer.h"
#include "file_read.h"


/*
Constructor
*/
BitsEnumerateCUDA::BitsEnumerateCUDA(string bedAFile,
							 string bedBFile,
							 string genomeFile) {

    _bedAFile = bedAFile;
    _bedBFile = bedBFile;
    _genomeFile = genomeFile;
    
    // create new BED file objects for A and B
    _bedA = new BedFile(bedAFile);
    _bedB = new BedFile(bedBFile);
    //_genome = new BedFile(genomeFile);
    _genome = new GenomeFile(genomeFile);
    
    EnumerateOverlapsCUDA();
}


/*
Destructor
*/
BitsEnumerateCUDA::~BitsEnumerateCUDA(void) {
}





void BitsEnumerateCUDA::EnumerateOverlapsCUDA()
{
	int *prt;
	hipMalloc(&prt, 0);

	vector<struct interval> A, B;
	read_and_map_files(_genome, &_offsets, _bedA, _bedB, &A, &B);
    										     
	unsigned int *R = (unsigned int *) malloc (sizeof(unsigned int) * A.size());
	unsigned int *E;

    uint32_t tot_overlaps = enumerate_intersections_bsearch_cuda(&A[0],
															 A.size(),
															 &B[0],
															 B.size(),
															 R,
															 &E);
	unsigned int start = 0;
	
    for (size_t i = 0; i < _bedA->bedList.size(); ++i) {
		while (start < R[i]) {
			printf("%s\t%d\t%d\t%s\t%d\t%d\n",
					_bedA->bedList[i].chrom.c_str(),
					_bedA->bedList[i].start,
					_bedA->bedList[i].end,
					_bedB->bedList[ E[start] ].chrom.c_str(),
					_bedB->bedList[ E[start] ].start,
					_bedB->bedList[ E[start] ].end);	
			++start;
		}
	}
}
