/*****************************************************************************
bits_count_cuda.cu
(c) 2012 - Ryan M. Layer
Hall Laboratory
Quinlan Laboratory
Department of Computer Science
Department of Biochemistry and Molecular Genetics
Department of Public Health Sciences and Center for Public Health Genomics,
University of Virginia
rl6sf@virginia.edu

Licenced under the GNU General Public License 2.0 license.
******************************************************************************/
#include "lineFileUtilities.h"
#include "bits_count_cuda.h"
#include "interval_cuda.h"
#include "timer.h"
#include "file_read.h"
#include "cutil.h"


/*
Constructor
*/
BitsCountCUDA::BitsCountCUDA(string bedAFile,
							 string bedBFile,
							 string genomeFile) {

    _bedAFile = bedAFile;
    _bedBFile = bedBFile;
    _genomeFile = genomeFile;
    
    // create new BED file objects for A and B
    _bedA = new BedFile(bedAFile);
    _bedB = new BedFile(bedBFile);
    //_genome = new BedFile(genomeFile);
    _genome = new GenomeFile(genomeFile);
    
    CountOverlapsCUDA();
}


/*
Destructor
*/
BitsCountCUDA::~BitsCountCUDA(void) {
}

void BitsCountCUDA::CountOverlapsCUDA() {
	int *prt;
	hipMalloc(&prt, 0);
	vector<struct interval> A, B;
	read_and_map_files(_genome, &_offsets, _bedA, _bedB, &A, &B);

    uint32_t tot_overlaps = count_intersections_bsearch_cuda(&A[0],
															 A.size(),
															 &B[0],
															 B.size());


	cout << tot_overlaps << endl;
}
