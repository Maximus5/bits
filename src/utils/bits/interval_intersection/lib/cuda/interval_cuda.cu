#include "hip/hip_runtime.h"
/*****************************************************************************
interval_cuda.cu
(c) 2012 - Ryan M. Layer
Hall Laboratory
Quinlan Laboratory
Department of Computer Science
Department of Biochemistry and Molecular Genetics
Department of Public Health Sciences and Center for Public Health Genomics,
University of Virginia
rl6sf@virginia.edu

Licenced under the GNU General Public License 2.0 license.
******************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include "cudpp.h"
#include "interval.h"
#include "timer.h"
#include "interval_cuda.h"
#include "bsearch_cuda.h"
#include "bsearch_cuda.cu"
#include <gsl/gsl_statistics_int.h>

//{{{ void per_interval_count_intersections_bsearch_cuda(struct interval *A,
void per_interval_count_intersections_bsearch_cuda(struct interval *A,
												  unsigned int size_A,
												  struct interval *B,
												  unsigned int size_B,
												  unsigned int *R)
{
	hipFree(NULL);

	unsigned int *A_starts_h, *A_lens_h, *B_starts_h, *B_ends_h;
	unsigned int *A_starts_d, *A_lens_d, *B_starts_d, *B_ends_d;
	unsigned int *R_d;
	allocate_and_move(A,
					 &A_starts_h,
					 &A_starts_d,
					 &A_lens_h ,
					 &A_lens_d,
					 size_A,
					 B,
					 &B_starts_h ,
					 &B_starts_d,
					 &B_ends_h ,
					 &B_ends_d,
					 size_B,
					 &R_d);

	cudpp_sort_by_key(B_starts_d, size_B);

	cudpp_sort_by_key(B_ends_d, size_B);

	bits_cuda(256,
			  1,
			  A_starts_d,
			  A_lens_d,
			  size_A,
			  B_starts_d,
			  B_ends_d,
			  size_B,
			  R_d);

	hipMemcpy(R, R_d, size_A* sizeof(unsigned int), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Result move: %s.\n", hipGetErrorString( err) );


	hipFree(A_starts_d);
	hipFree(A_lens_d);
	hipFree(B_starts_d);
	hipFree(B_ends_d);
	hipFree(R_d);
	free(A_starts_h);
	free(A_lens_h);
	free(B_starts_h);
	free(B_ends_h);
}
//}}}

//{{{ unsigned int count_intersections_bsearch_cuda(struct interval *A,
unsigned int count_intersections_bsearch_cuda(struct interval *A,
										      unsigned int size_A,
											  struct interval *B,
										      unsigned int size_B)
{
	//hipFree(NULL);

	unsigned int *A_starts_h, *A_lens_h, *B_starts_h, *B_ends_h;
	unsigned int *A_starts_d, *A_lens_d, *B_starts_d, *B_ends_d;
	unsigned int *R_d;
	allocate_and_move(A,
					 &A_starts_h,
					 &A_starts_d,
					 &A_lens_h ,
					 &A_lens_d,
					 size_A,
					 B,
					 &B_starts_h ,
					 &B_starts_d,
					 &B_ends_h ,
					 &B_ends_d,
					 size_B,
					 &R_d);

	cudpp_sort_by_key(B_starts_d, size_B);

	cudpp_sort_by_key(B_ends_d, size_B);

	bits_cuda(256,
			  1,
			  A_starts_d,
			  A_lens_d,
			  size_A,
			  B_starts_d,
			  B_ends_d,
			  size_B,
			  R_d);

	unsigned int *Ro_d;
	hipMalloc((void **)&Ro_d, sizeof(unsigned int));
	hipMemset(Ro_d, 0, sizeof(unsigned int));

	cudpp_sum(R_d, Ro_d, size_A);

	unsigned int R;
	hipMemcpy(&R, Ro_d, sizeof(unsigned int), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Result move: %s.\n", hipGetErrorString( err) );

	hipFree(A_starts_d);
	hipFree(A_lens_d);
	hipFree(B_starts_d);
	hipFree(B_ends_d);
	hipFree(R_d);
	hipFree(Ro_d);
	free(A_starts_h);
	free(A_lens_h);
	free(B_starts_h);
	free(B_ends_h);

	return R;
}
//}}}

//{{{ unsigned int test_intersections_bsearch_cuda(struct interval *A,
unsigned int test_intersections_bsearch_cuda(struct interval *A,
										     unsigned int size_A,
											 struct interval *B,
										     unsigned int size_B,
											 unsigned int n,
											 unsigned int max_offset,
											 unsigned int *O,
											 double *mean,
											 double *sd,
											 double *p)
{
	hipError_t err;
	int block_size = 256;
	dim3 dimBlock(block_size);
	int grid_size;

	//{{{ allocate and move A_starts, A_lens, B_starts, B_ends, B_lens
	struct timeval t_start = in();
	unsigned int *A_starts_h, *A_lens_h, *B_starts_h, *B_ends_h;
	unsigned int *A_starts_d, *A_lens_d, *B_starts_d, *B_ends_d;
	unsigned int *R_d;
	struct timeval start = in();
	allocate_and_move(A,
					&A_starts_h,
					&A_starts_d,
					&A_lens_h ,
					&A_lens_d,
					size_A,
					B,
					&B_starts_h ,
					&B_starts_d,
					&B_ends_h ,
					&B_ends_d,
					size_B,
					&R_d);
	unsigned int *Ro_d;
	hipMalloc((void **)&Ro_d, sizeof(unsigned int));
	hipMemset(Ro_d, 0, sizeof(unsigned int));
	unsigned int *B_lens_d;
	hipMalloc((void **)&B_lens_d, size_B*sizeof(unsigned int));

	unsigned int i_mem_time = out(t_start);
	//fprintf(stderr, "i_mem:%u\t", i_mem_time);
	//}}}
	
	cudpp_sort_by_key_value( A_starts_d,
							 A_lens_d,
							 size_A);

	cudpp_sort_by_key_value( B_starts_d,
							 B_ends_d,
							 size_B);

	//{{{ set B_len, ordered by B_start
	grid_size = ( size_B + block_size - 1) / (block_size * 1);
	dim3 dimGridSet( grid_size );
	set_len_cuda <<<dimGridSet, dimBlock >>> (B_starts_d,
											  B_ends_d,
											  B_lens_d,
											  size_B,
											  1);
	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "set_len_cuda: %s.\n", hipGetErrorString( err) );

	//}}}

	cudpp_sort_by_key(B_ends_d,
					  size_B);

	bits_cuda(256,
			  1,
			  A_starts_d,
			  A_lens_d,
			  size_A,
			  B_starts_d,
			  B_ends_d,
			  size_B,
			  R_d);
	
	unsigned int i_do_time = out(t_start);
	cudpp_sum(R_d, Ro_d, size_A);


	//{{{ Move R_d to R
	t_start = in();
	unsigned int R;
	hipMemcpy(&R, Ro_d, sizeof(unsigned int), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Result move: %s.\n", hipGetErrorString( err) );

	i_mem_time += out(t_start);
	//fprintf(stderr, "i_mem:%u\t", i_mem_time);
	//}}}
	
	*O = R;

	// Generate random A_start and B_start
	srand(1);
	unsigned int *sims = (unsigned int *) malloc(n * sizeof(unsigned int));
	int i, N = 0;
	for(i = 0; i < n; i++) {

		//{{{  Rands for A_start and B_start
		cudpp_rand(A_starts_d, rand(), size_A);
		cudpp_rand(B_starts_d, rand(), size_B);

		unsigned int shift = 4294967295;

		grid_size = ( size_B + block_size - 1) / (block_size * 1);
		dim3 dimGridMapB( grid_size );
		map_list_cuda <<<dimGridMapB, dimBlock >>> (B_starts_d,
													size_B,
													shift,
													max_offset,
													1);
		hipDeviceSynchronize();
		err = hipGetLastError();
		if(err != hipSuccess)
			fprintf(stderr, "map_list_cuda B: %s.\n", hipGetErrorString( err) );

		grid_size = ( size_A + block_size - 1) / (block_size * 1);
		dim3 dimGridMapA( grid_size );
		map_list_cuda <<<dimGridMapA, dimBlock >>> (A_starts_d,
													size_A,
													shift,
													max_offset,
													1);
		hipDeviceSynchronize();
		err = hipGetLastError();
		if(err != hipSuccess)
			fprintf(stderr, "map_list_cuda A: %s.\n", hipGetErrorString( err) );
		//}}}
		
		cudpp_sort_by_key(A_starts_d, size_A);
		cudpp_sort_by_key(B_starts_d, size_B);
			
		//{{{ set B_end, ordered by B_start
		grid_size = ( size_B + block_size - 1) / (block_size * 1);
		//dimGridSet( grid_size );
		set_end_cuda <<<dimGridSet, dimBlock >>> (B_starts_d,
												  B_ends_d,
												  B_lens_d,
												  size_B,
												  1);
		hipDeviceSynchronize();
		err = hipGetLastError();
		if(err != hipSuccess)
			fprintf(stderr, "map_list_cuda A: %s.\n", hipGetErrorString( err) );

		//}}}

		bits_cuda(256,
				  1,
				  A_starts_d,
				  A_lens_d,
				  size_A,
				  B_starts_d,
				  B_ends_d,
				  size_B,
				  R_d);

		unsigned int i_do_time = out(t_start);
		cudpp_sum(R_d, Ro_d, size_A);

		//{{{ Move R_d to R
		t_start = in();
		hipMemcpy(&R, Ro_d, sizeof(unsigned int), hipMemcpyDeviceToHost);

		hipDeviceSynchronize();
		err = hipGetLastError();
		if(err != hipSuccess)
			fprintf(stderr, "Result move: %s.\n", hipGetErrorString( err) );

		i_mem_time += out(t_start);
		//fprintf(stderr, "i_mem:%u\t", i_mem_time);
		//}}}
		
		sims[i] = R;	
	
		if (R >= *O)
			N = N + 1;
	}

	*mean = gsl_stats_int_mean((const int*)sims, 1, n);
	*sd = gsl_stats_int_sd_m((const int*)sims, 1, n, *mean);
	*p =  ( (double) N + 1) / ( (double) n + 1);

	hipFree(A_starts_d);
	hipFree(A_lens_d);
	hipFree(B_starts_d);
	hipFree(B_ends_d);
	hipFree(B_lens_d);
	hipFree(R_d);
	hipFree(Ro_d);
	free(A_starts_h);
	free(A_lens_h);
	free(B_starts_h);
	free(B_ends_h);
	free(sims);
	
	return 0;
}
//}}}

//{{{ unsigned int enumerate_intersections_bsearch_cuda(struct interval *A,
unsigned int enumerate_intersections_bsearch_cuda(struct interval *A,
										      unsigned int size_A,
											  struct interval *B,
										      unsigned int size_B,
											  unsigned int *R,
											  unsigned int **E)
{
	hipFree(NULL);

	unsigned int *A_starts_h, *A_lens_h, *B_starts_h, *B_ends_h;
	unsigned int *A_starts_d, *A_lens_d, *B_starts_d, *B_ends_d;
	unsigned int *R_d;

	allocate_and_move(A,
					 &A_starts_h,
					 &A_starts_d,
					 &A_lens_h ,
					 &A_lens_d,
					 size_A,
					 B,
					 &B_starts_h ,
					 &B_starts_d,
					 &B_ends_h ,
					 &B_ends_d,
					 size_B,
					 &R_d);

	cudpp_sort_by_key(B_ends_d, size_B);

	// B_starts_id is the index of the interval in the orginial bed file, when
	// the intersecting intervals are enumerated, the B_starts_id value will be
	// stored so that the correct intervals can be displayed 
	unsigned int *B_starts_id_d;
	hipMalloc((void **)&B_starts_id_d, size_B*sizeof(unsigned int));

	int block_size = 256;
	dim3 dimBlock(block_size);
	int grid_size = ( size_B + block_size - 1) / (block_size * 1);
	dim3 dimGridIdSet( grid_size );

	set_id_cuda <<<dimGridIdSet, dimBlock>>> (B_starts_id_d, size_B, 1);

	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "set_id_cuda: %s.\n", hipGetErrorString( err) );

	cudpp_sort_by_key_value(B_starts_d, B_starts_id_d, size_B);

	bits_cuda(256,
			  1,
			  A_starts_d,
			  A_lens_d,
			  size_A,
			  B_starts_d,
			  B_ends_d,
			  size_B,
			  R_d);

	unsigned int *Ro_d;
	hipMalloc((void **)&Ro_d, size_A*sizeof(unsigned int));

	// Do a prefix scan of the results to be used to identify the offset of the
	// intervals within B
	cudpp_scan(R_d, Ro_d, size_A);

	hipMemcpy(R, Ro_d , size_A*sizeof(unsigned int), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Result move: %s.\n", hipGetErrorString( err) );
	
	unsigned int N = (R)[size_A - 1];

	hipFree(R_d);

	hipMemcpy(B_starts_d, B_starts_h, size_B * sizeof(unsigned int),
			hipMemcpyHostToDevice);
	hipMemcpy(B_ends_d, B_ends_h, size_B * sizeof(unsigned int),
			hipMemcpyHostToDevice);

	cudpp_sort_by_key_value( B_starts_d,
							 B_ends_d,
							 size_B);

	unsigned int *E_d;
	hipMalloc((void **)&E_d, N*sizeof(unsigned int));

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Malloc E: %s.\n", hipGetErrorString( err) );

	grid_size = ( size_A + block_size - 1) / (block_size * 1);
	dim3 dimGridSearch( grid_size );

	enumerate_bsearch_cuda <<< dimGridSearch, dimBlock >>> (
			A_starts_d, A_lens_d, size_A,
			B_starts_d, B_ends_d, size_B,
			B_starts_id_d,
			Ro_d,
			E_d,
			1);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "enumerate_bsearch_cuda: %s.\n",
					hipGetErrorString( err) );

	*E = (unsigned int *) malloc( N*sizeof(unsigned int) );
	hipMemcpy(*E, E_d, N*sizeof(unsigned int), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Result move: %s.\n", hipGetErrorString( err) );
	return N;
}
//}}}

//{{{void cudpp_rand_init(CUDPPHandle *rand_cudpp,
void cudpp_rand_init(CUDPPHandle *rand_cudpp,
					 CUDPPHandle *rand_plan,
					 unsigned int seed,
					 unsigned int size)
{
	cudppCreate(rand_cudpp);

	CUDPPConfiguration rand_config;
	rand_config.op = CUDPP_ADD;
	rand_config.datatype = CUDPP_UINT;
	rand_config.algorithm = CUDPP_RAND_MD5;
	rand_config.options = 0;

	CUDPPResult res = cudppPlan(*rand_cudpp, 
								rand_plan,
								rand_config,
								size,
								1,
								0);;
	if (CUDPP_SUCCESS != res) {
		printf("Error creating rand CUDPPPlan\n");
        exit(-1);
	}

	res = cudppRandSeed(*rand_plan, seed);
	if (CUDPP_SUCCESS != res) {
		printf("Error in cudppRandSeed\n");
        exit(-1);
	}

}
//}}}	

//{{{ void cudpp_rand(unsigned int *keys_d,
void cudpp_planned_rand(CUDPPHandle *rand_plan,
						unsigned int *out_d,
						unsigned int size)
{

	CUDPPResult res = cudppRand(*rand_plan, out_d, size);
    if (CUDPP_SUCCESS != res) {
        printf("Error in cudppRand()\n");
        exit(-1);
    }
}
//}}}

//{{{ void cudpp_rand(unsigned int *keys_d,
void cudpp_rand(unsigned int *out_d,
				unsigned int seed,
				unsigned int size)
{

	CUDPPHandle rand_cudpp;
	cudppCreate(&rand_cudpp);

	CUDPPConfiguration rand_config;
	rand_config.op = CUDPP_ADD;
	rand_config.datatype = CUDPP_UINT;
	rand_config.algorithm = CUDPP_RAND_MD5;
	rand_config.options = 0;

	CUDPPHandle rand_plan= 0;
	CUDPPResult res = cudppPlan(rand_cudpp, 
								&rand_plan,
								rand_config,
								size,
								1,
								0);;

	if (CUDPP_SUCCESS != res) {
		printf("Error creating rand CUDPPPlan\n");
        exit(-1);
	}

	res = cudppRandSeed(rand_plan, seed);
	if (CUDPP_SUCCESS != res) {
		printf("Error in cudppRandSeed\n");
        exit(-1);
	}

	res = cudppRand(rand_plan, out_d, size);
    if (CUDPP_SUCCESS != res) {
        printf("Error in cudppRand()\n");
        exit(-1);
    }

	res = cudppDestroy(rand_cudpp);
    if (CUDPP_SUCCESS != res) {
		printf("Error shutting down CUDPP Library.\n");
        exit(-1);
    }
}
//}}}

//{{{void cudpp_sort_by_key( unsigned int *keys_d,
void cudpp_sort_by_key( unsigned int *keys_d,
						unsigned int size)
{
	CUDPPHandle sort_cudpp;
	cudppCreate(&sort_cudpp);

	CUDPPConfiguration sort_config;
	sort_config.datatype = CUDPP_UINT;
	sort_config.algorithm = CUDPP_SORT_RADIX;
	sort_config.options = CUDPP_OPTION_KEYS_ONLY;

	CUDPPHandle sort_plan= 0;
	CUDPPResult res = cudppPlan(sort_cudpp, 
								&sort_plan,
								sort_config,
								size,
								1,
								0);;

	if (CUDPP_SUCCESS != res) {
		printf("Error creating sort CUDPPPlan\n");
        exit(-1);
	}

    res = cudppSort(sort_plan, keys_d, NULL, size);
    if (CUDPP_SUCCESS != res) {
        printf("Error in cudppSort()\n");
        exit(-1);
    }

	res = cudppDestroy(sort_cudpp);
    if (CUDPP_SUCCESS != res) {
		printf("Error shutting down CUDPP Library.\n");
        exit(-1);
    }
}
//}}}

//{{{void cudpp_sort_by_key_value( unsigned int *keys_d,
void cudpp_sort_by_key_value(unsigned int *keys_d,
							 unsigned int *values_d,
							 unsigned int size)
{
	CUDPPHandle sort_cudpp;
	cudppCreate(&sort_cudpp);

	CUDPPConfiguration sort_config;
	sort_config.datatype = CUDPP_UINT;
	sort_config.algorithm = CUDPP_SORT_RADIX;
	sort_config.options = CUDPP_OPTION_KEY_VALUE_PAIRS;

	CUDPPHandle sort_plan= 0;
	CUDPPResult res = cudppPlan(sort_cudpp, 
								&sort_plan,
								sort_config,
								size,
								1,
								0);;

	if (CUDPP_SUCCESS != res) {
		printf("Error creating sort CUDPPPlan\n");
        exit(-1);
	}

    res = cudppSort(sort_plan, keys_d, values_d, size);
    if (CUDPP_SUCCESS != res) {
        printf("Error in cudppSort()\n");
        exit(-1);
    }

	res = cudppDestroy(sort_cudpp);
    if (CUDPP_SUCCESS != res) {
		printf("Error shutting down CUDPP Library.\n");
        exit(-1);
    }
}
//}}}

//{{{ void cudpp_sum(unsigned int *list_d,
void cudpp_sum(unsigned int *list_d,
			   unsigned int *sum_d,
			   unsigned int size)
{
	CUDPPHandle sum_cudpp;
	cudppCreate(&sum_cudpp);

	CUDPPConfiguration sum_config;
	sum_config.datatype = CUDPP_UINT;
	sum_config.algorithm = CUDPP_REDUCE;
	sum_config.options = 0;

	CUDPPHandle sum_plan = 0;
	CUDPPResult res = cudppPlan(sum_cudpp, 
								&sum_plan,
								sum_config,
								size,
								1,
								0);

	if (CUDPP_SUCCESS != res) {
		printf("Error creating sort CUDPPPlan\n");
        exit(-1);
	}

    res = cudppReduce(sum_plan, sum_d, list_d, size);
    if (CUDPP_SUCCESS != res) {
        printf("Error in cudppReduce() R_d\n");
        exit(-1);
    }

	res = cudppDestroy(sum_plan);
    if (CUDPP_SUCCESS != res) {
		printf("Error shutting down CUDPP Library.\n");
        exit(-1);
    }
}
//}}}

//{{{void cudpp_scan(unsigned int *R_d,
void cudpp_scan(unsigned int *R_d,
			    unsigned int *Ro_d,
			    unsigned int size)
{
	CUDPPHandle scan_cudpp;
	cudppCreate(&scan_cudpp);

	CUDPPConfiguration scan_config;
	scan_config.op = CUDPP_ADD;
	scan_config.datatype = CUDPP_UINT;
	scan_config.algorithm = CUDPP_SCAN;
	scan_config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;

	//unsigned int *Ro_d;
	//hipMalloc((void **)&Ro_d, size_A*sizeof(unsigned int));

	CUDPPHandle scan_plan = 0;
	CUDPPResult res = cudppPlan(scan_cudpp, 
								&scan_plan,
								scan_config,
								size,
								1,
								0);

	if (CUDPP_SUCCESS != res) {
		printf("Error creating scan CUDPPPlan\n");
        exit(-1);
	}

    res = cudppScan(scan_plan, Ro_d, R_d, size);
    if (CUDPP_SUCCESS != res) {
        printf("Error in cudppScan() R_d\n");
        exit(-1);
    }

	res = cudppDestroy(scan_plan);
    if (CUDPP_SUCCESS != res) {
		printf("Error shutting down CUDPP Library.\n");
        exit(-1);
    }
}
//}}}	

//{{{ unsigned int count_intersections_sort_bsearch_cuda(struct interval *A,
unsigned int count_intersections_sort_bsearch_cuda(struct interval *A,
										      unsigned int size_A,
											  struct interval *B,
										      unsigned int size_B)
{
	int block_size = 256;
	dim3 dimBlock(block_size);
	int grid_size = ( size_A + block_size - 1) / (block_size * 1);
	dim3 dimGridSearch( grid_size );
	hipError_t err;

	start(); //data_prep_time
	//{{{ Allocate and move 
	unsigned int *A_starts_h, *A_lens_h, *B_starts_h, *B_ends_h;
	unsigned int *A_starts_d, *A_lens_d, *B_starts_d, *B_ends_d;
	unsigned int *R_d;
	allocate_and_move(A,
					&A_starts_h,
					&A_starts_d,
					&A_lens_h ,
					&A_lens_d,
					size_A,
					B,
					&B_starts_h ,
					&B_starts_d,
					&B_ends_h ,
					&B_ends_d,
					size_B,
					&R_d);
	//}}}
	stop(); //data_prep_time
	unsigned long data_prep_time = report();

	start(); //sort_time
	//{{{ Sort B_starts and B_ends
	// Sort B by start
	//nvRadixSort::RadixSort radixsortB_starts(size_B, true);
	//radixsortB_starts.sort((unsigned int*)B_starts_d, 0, size_B, 32);

	//hipDeviceSynchronize();
	//err = hipGetLastError();
	//if(err != hipSuccess)
		//fprintf(stderr, "Sort B_starts: %s.\n", hipGetErrorString( err) );

	// Sort B by end
	//nvRadixSort::RadixSort radixsortB_ends(size_B, true);
	//radixsortB_ends.sort((unsigned int*)B_ends_d, 0, size_B, 32);

	//hipDeviceSynchronize();
	//err = hipGetLastError();
	//if(err != hipSuccess)
		//fprintf(stderr, "Sort B_ends: %s.\n", hipGetErrorString( err) );
	//}}}
	stop();	//sort_time
	unsigned long sort_time = report();
	
	start();
	//{{{ Compute and count intersections
	count_bsearch_cuda <<<dimGridSearch, dimBlock >>> (
			A_starts_d, A_lens_d, size_A,
			B_starts_d, B_ends_d, size_B,
			R_d,
			1);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Sort B_ends: %s.\n", hipGetErrorString( err) );

	//parallel_sum(R_d, block_size, size_A, 1024);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Parallel sum: %s.\n", hipGetErrorString( err) );


	unsigned int R;
	hipMemcpy(&R, R_d, sizeof(unsigned int), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Result move: %s.\n", hipGetErrorString( err) );

	//}}}
	stop(); //intersect_time
	/*
	unsigned long intersect_time = report();

	unsigned long total_time = data_prep_time + 
							   sort_time +
							   pre_sort_time +
							   intersect_time;
	printf("sort\t"
		   "total:%lu\t"
		   "prep:%lu,%f\t"
		   "sort:%lu,%f\t"
		   "presort:%lu,%f\t"
		   "intersect:%lu,%f\n",
		   total_time,
		   data_prep_time,  (double)data_prep_time / (double)total_time,
		   sort_time, (double)sort_time / (double)total_time,
		   pre_sort_time, (double)pre_sort_time / (double)total_time,
		   intersect_time, (double)intersect_time / (double)total_time);
	*/
	hipFree(A_starts_d);
	hipFree(A_lens_d);
	hipFree(B_starts_d);
	hipFree(B_ends_d);
	hipFree(R_d);

	return R;
}
//}}}

//{{{ unsigned int count_intersections_i_bsearch_cuda(struct interval *A,
unsigned int count_intersections_i_gm_bsearch_cuda(struct interval *A,
										      unsigned int size_A,
											  struct interval *B,
										      unsigned int size_B,
										      unsigned int size_I)
{
	int block_size = 256;
	dim3 dimBlock(block_size);
	int grid_size = ( size_A + block_size - 1) / (block_size);
	dim3 dimGridSearch( grid_size );
	hipError_t err;

	start(); //data_prep_time

	//{{{ Allocate and move 
	unsigned int *A_starts_h, *A_lens_h, *B_starts_h, *B_ends_h;
	unsigned int *A_starts_d, *A_lens_d, *B_starts_d, *B_ends_d;
	unsigned int *R_d;
	allocate_and_move(A,
					&A_starts_h,
					&A_starts_d,
					&A_lens_h ,
					&A_lens_d,
					size_A,
					B,
					&B_starts_h ,
					&B_starts_d,
					&B_ends_h ,
					&B_ends_d,
					size_B,
					&R_d);
	//}}}
	//
	stop(); //data_prep_time
	unsigned long data_prep_time = report();

	start();//sort_time
	//{{{ Sort B_starts and B_ends
	// Sort B by start
	//nvRadixSort::RadixSort radixsortB_starts(size_B, true);
	//radixsortB_starts.sort((unsigned int*)B_starts_d, 0, size_B, 32);

	//hipDeviceSynchronize();
	//err = hipGetLastError();
	//if(err != hipSuccess)
		//fprintf(stderr, "Sort B_starts: %s.\n", hipGetErrorString( err) );

	// Sort B by end
	//nvRadixSort::RadixSort radixsortB_ends(size_B, true);
	//radixsortB_ends.sort((unsigned int*)B_ends_d, 0, size_B, 32);

	//hipDeviceSynchronize();
	//err = hipGetLastError();
	//if(err != hipSuccess)
		//fprintf(stderr, "Sort B_ends: %s.\n", hipGetErrorString( err) );
	//}}}
	stop();	//sort_time
	unsigned long sort_time = report();

	start();//index_time
	//{{{ Generate index
	unsigned int *I_starts_d, *I_ends_d;
	hipMalloc((void **)&I_starts_d, (size_I)*sizeof(unsigned int));
	hipMalloc((void **)&I_ends_d, (size_I)*sizeof(unsigned int));

	int index_grid_size = ( size_I + block_size - 1) / (block_size);
	dim3 index_dimGrid( index_grid_size );

	gen_index <<<index_dimGrid, dimBlock>>> ( B_starts_d, size_B, I_starts_d, size_I);
	gen_index <<<index_dimGrid, dimBlock>>> ( B_ends_d, size_B, I_ends_d, size_I);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Count i bsearch: %s.\n", hipGetErrorString( err) );
	//}}}
	stop();	//index_time
	unsigned long index_time = report();

	//{{{ Compute and count intersections
	count_i_gm_bsearch_cuda <<<dimGridSearch, dimBlock >>> (
			A_starts_d, A_lens_d, size_A,
			B_starts_d, B_ends_d, size_B,
			I_starts_d, I_ends_d, size_I,
			R_d,
			1);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Count i bsearch: %s.\n", hipGetErrorString( err) );

	//parallel_sum(R_d, block_size, size_A, 1024);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Parallel sum: %s.\n", hipGetErrorString( err) );


	unsigned int R;
	hipMemcpy(&R, R_d, sizeof(unsigned int), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Result move: %s.\n", hipGetErrorString( err) );

	//}}}
	stop(); //intersect_time
	unsigned long intersect_time = report();

	unsigned long total_time = data_prep_time + 
							   sort_time +
							   index_time +
							   intersect_time;
	printf("index gm\t"
		   "total:%lu\t"
		   "prep:%lu,%f\t"
		   "sort:%lu,%f\t"
		   "index:%lu,%f\t"
		   "intersect:%lu,%f\n",
		   total_time,
		   data_prep_time,  (double)data_prep_time / (double)total_time,
		   sort_time, (double)sort_time / (double)total_time,
		   index_time, (double)index_time / (double)total_time,
		   intersect_time, (double)intersect_time / (double)total_time);

	hipFree(A_starts_d);
	hipFree(A_lens_d);
	hipFree(B_starts_d);
	hipFree(B_ends_d);
	hipFree(I_starts_d);
	hipFree(I_ends_d);
	hipFree(R_d);

	return R;
}
//}}}

//{{{ __global__ void count_bsearch_cuda (	unsigned int *A_start,
/*
 * @param A_start list of start positions to query, does not need to be sorted
 * @param A_len list of lengths that correspond to A_start
 * @param A_size size of A_start and A_len
 * @param B_start list of sorted start positions to be queried
 * @param B_end list of sorted end positions to be queired 
 * @param B_size size of B_start and B_end
 * @param R number of intersections for each interval in A
 * @param n number of intervals per thread
 */
__global__
void count_bsearch_cuda (	unsigned int *A_start,
							unsigned int *A_len,
							int A_size,
							unsigned int *B_start,
							unsigned int *B_end,
							int B_size,
							unsigned int *R,
							int n)
{
	unsigned int id = (blockIdx.x * blockDim.x) + threadIdx.x;

	unsigned int i = id;
	unsigned int grid_size = blockDim.x * gridDim.x;

	while ( i < (n * grid_size) ) {

		if (i < A_size) {
			unsigned int start = A_start[i];
			unsigned int end = start + A_len[i];

			int cant_before = bound_binary_search(B_end,
														   B_size,
														   start,
														   -1,
														   B_size);

			int cant_after = bound_binary_search(B_start,
														  B_size,
														  end,
														  -1,
														  B_size);

			while ( end == B_start[cant_after] )
				++cant_after;

			cant_after = A_size - cant_after;	

			R[i] = A_size - cant_before - cant_after;
		}
		i += grid_size;
	}
}
//}}}

//{{{ __global__ void count_i_gm_bsearch_cuda (	unsigned int *A_start,
/*
 * @param A_start list of start positions to query, does not need to be sorted
 * @param A_len list of lengths that correspond to A_start
 * @param A_size size of A_start and A_len
 * @param B_start list of sorted start positions to be queried
 * @param B_end list of sorted end positions to be queired 
 * @param B_size size of B_start and B_end
 * @param R number of intersections for each interval in A
 * @param n number of intervals per thread
 */
__global__
void count_i_gm_bsearch_cuda (	unsigned int *A_start,
							unsigned int *A_len,
							int A_size,
							unsigned int *B_start,
							unsigned int *B_end,
							int B_size,
							unsigned int *I_start,
							unsigned int *I_end,
							int I_size,
							unsigned int *R,
							int n)
{
	unsigned int id = (blockIdx.x * blockDim.x) + threadIdx.x;

	unsigned int i = id;
	unsigned int grid_size = blockDim.x * gridDim.x;

	while ( i < (n * grid_size) ) {
		if (i < A_size) {
			unsigned int start = A_start[i];
			unsigned int end = start + A_len[i];

			int cant_before = i_binary_search(B_end,
											  B_size,
											  start,
											  I_end,
											  I_size);
	
			int cant_after = i_binary_search(B_start,
											 B_size,
											 end,
											 I_start,
											 I_size);

			while ( end == B_start[cant_after] )
				++cant_after;

			cant_after = A_size - cant_after;	

			R[i] = A_size - cant_before - cant_after;
		}
		i += grid_size;
	}
}
//}}}

//{{{void allocate_and_move( struct interval *A,
void allocate_and_move( struct interval *A,
						unsigned int **A_starts_h,
						unsigned int **A_starts_d,
					   	unsigned int **A_lens_h ,
						unsigned int **A_lens_d,
						unsigned int size_A,

						struct interval *B,
						unsigned int **B_starts_h ,
						unsigned int **B_starts_d,
						unsigned int **B_ends_h ,
						unsigned int **B_ends_d,
						unsigned int size_B,

						unsigned int **R_d)
{

	hipError_t err;
	//{{{ Move intervals to unsigned int arrays
	*A_starts_h = (unsigned int *) malloc( (size_A) * sizeof(unsigned int));
	*A_lens_h = (unsigned int *) malloc( (size_A) * sizeof(unsigned int));

	*B_starts_h = (unsigned int *) malloc( (size_B) * sizeof(unsigned int));
	*B_ends_h = (unsigned int *) malloc( (size_B) * sizeof(unsigned int));

	int i;
	for (i = 0; i < size_B; i++) {
		(*B_starts_h)[i] = B[i].start;
		(*B_ends_h)[i] = B[i].end;
	}

	for (i = 0; i < size_A; i++) {
		(*A_starts_h)[i] = A[i].start;
		(*A_lens_h)[i] = A[i].end - A[i].start;
	}
	//}}}

	//{{{ Move inteval arrays to device
	hipMalloc((void **)A_starts_d, (size_A)*sizeof(unsigned int));
	hipMalloc((void **)A_lens_d, (size_A)*sizeof(unsigned int));
	hipMalloc((void **)B_starts_d, (size_B)*sizeof(unsigned int));
	hipMalloc((void **)B_ends_d, (size_B)*sizeof(unsigned int));

	hipMemcpy(*A_starts_d, *A_starts_h, (size_A) * sizeof(unsigned int), 
			hipMemcpyHostToDevice);
	hipMemcpy(*A_lens_d, *A_lens_h, (size_A) * sizeof(unsigned int),
			hipMemcpyHostToDevice);
	hipMemcpy(*B_starts_d, *B_starts_h, (size_B) * sizeof(unsigned int), 
			hipMemcpyHostToDevice);
	hipMemcpy(*B_ends_d, *B_ends_h, (size_B) * sizeof(unsigned int),
			hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Interval move: %s.\n", hipGetErrorString( err) );
	//}}}
	
	//{{{ Alocate space for result on device
	hipMalloc((void **)R_d, (size_A)*sizeof(unsigned int));
	unsigned long memup_time = report();

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "R_d malloc: %s.\n", hipGetErrorString( err) );
	//}}}
}
//}}}

//{{{ __global__ void enumerate_bsearch_cuda (	unsigned int *A_start,
/*
 * @param A_start list of start positions to query, does not need to be sorted
 * @param A_len list of lengths that correspond to A_start
 * @param A_size size of A_start and A_len
 * @param B_start list of sorted start positions to be queried
 * @param B_end list of sorted end positions to be queired 
 * @param B_size size of B_start and B_end
 * @param R number of intersections for each interval in A
 * @param n number of intervals per thread
 */
__global__
void enumerate_bsearch_cuda (unsigned int *A_start,
							 unsigned int *A_len,
							 int A_size,
							 unsigned int *B_start,
							 unsigned int *B_end,
							 int B_size,
							 unsigned int *B_starts_id_d,
							 unsigned int *R,
							 unsigned int *E,
							 int n)
{
	unsigned int id = (blockIdx.x * blockDim.x) + threadIdx.x;

	unsigned int i = id;
	unsigned int grid_size = blockDim.x * gridDim.x;

	while ( i < (n * grid_size) ) {
		if (i < A_size) {
			unsigned int A_i_start = A_start[i];
			unsigned int A_i_end = A_i_start + A_len[i];

			unsigned int start= 0, end;

			if (i != 0)
				start = R[i - 1];

			end = R[i];

			if (end - start > 0) {
				unsigned int from = bound_binary_search(B_start,
													    B_size,
													    A_i_end,
													    -1,
													    B_size);

				while ( ( B_start[from] == A_i_end) && (from < B_size))
					++from;

				while (  (end - start) > 0 ) {
					if ( (A_i_start <= B_end[from]) && 
							(A_i_end >= B_start[from]) ) {
						 E[start] = B_starts_id_d[from];
						 ++start;
					}

					--from;
				}
			}
		}
		i += grid_size;
	}
}
//}}}

//{{{ __global__ void set_len_cuda (	unsigned int *start,
__global__
void set_len_cuda (	unsigned int *start,
					unsigned int *end,
					unsigned int *len,
					int size,
					int n)
{
	unsigned int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int i = id;
	unsigned int grid_size = blockDim.x * gridDim.x;

	while ( i < (n * grid_size) ) {
		if (i < size) {
			len[i] = end[i] - start[i];
		}
		i += grid_size;
	}
}
//}}}

//{{{ __global__ void set_end_cuda (	unsigned int *start,
__global__
void set_end_cuda (	unsigned int *start,
					unsigned int *end,
					unsigned int *len,
					int size,
					int n)
{
	unsigned int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int i = id;
	unsigned int grid_size = blockDim.x * gridDim.x;

	while ( i < (n * grid_size) ) {
		if (i < size) {
			end[i] = start[i] + len[i];
		}
		i += grid_size;
	}
}
//}}}

//{{{ __global__ void map_list_cuda (unsigned int *list,
__global__
void map_list_cuda (unsigned int *list,
					unsigned int size,
					unsigned int old_max,
					unsigned int new_max,
					int n)
{
	unsigned int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int i = id;
	unsigned int grid_size = blockDim.x * gridDim.x;
	float norm = (float) new_max/(float) old_max;

	while ( i < (n * grid_size) ) {
		if (i < size) {
			list[i] = (unsigned int)(list[i] * norm);
		}
		i += grid_size;
	}
}
//}}}

//{{{ __global__ void set_len_cuda (	unsigned int *start,
__global__
void set_id_cuda (unsigned int *list,
				  int size,
				  int n)
{
	unsigned int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int i = id;
	unsigned int grid_size = blockDim.x * gridDim.x;

	while ( i < (n * grid_size) ) {
		if (i < size) {
			list[i] = i;
		}
		i += grid_size;
	}
}
//}}}

//{{{void bits_cuda(int block_size,
void bits_cuda(int block_size,
			   unsigned int per_thread,
			   unsigned int *A_starts_d,
			   unsigned int *A_lens_d,
			   unsigned int size_A,
			   unsigned int *B_starts_d,
			   unsigned int *B_ends_d,
			   unsigned int size_B,
			   unsigned int *R_d) 
{
	dim3 dimBlock(block_size);
	int grid_size = ( size_A + block_size - 1) / (block_size * 1);
	dim3 dimGridSearch( grid_size );

	count_bsearch_cuda <<< dimGridSearch, dimBlock >>> (
			A_starts_d, A_lens_d, size_A,
			B_starts_d, B_ends_d, size_B,
			R_d,
			1);

	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "count_bsearch_cuda: %s.\n", hipGetErrorString( err) );
}
//}}}

//{{{ void cuda_free() {
void cuda_free() {
	hipFree(NULL);
}
//}}}
