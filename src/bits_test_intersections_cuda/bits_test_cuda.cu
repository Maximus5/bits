/*****************************************************************************
bits_test_cuda.cu
(c) 2012 - Ryan M. Layer
Hall Laboratory
Quinlan Laboratory
Department of Computer Science
Department of Biochemistry and Molecular Genetics
Department of Public Health Sciences and Center for Public Health Genomics,
University of Virginia
rl6sf@virginia.edu

Licenced under the GNU General Public License 2.0 license.
******************************************************************************/
#include "lineFileUtilities.h"
#include "bits_test_cuda.h"
#include "interval_cuda.h"
#include "timer.h"
#include "file_read.h"


/*
Constructor
*/
BitsTestCUDA::BitsTestCUDA( string bedAFile,
							string bedBFile,
							string genomeFile,
							unsigned int N) {
    _bedAFile = bedAFile;
    _bedBFile = bedBFile;
    _genomeFile = genomeFile;
	_N = N;
    
    // create new BED file objects for A and B
    _bedA = new BedFile(bedAFile);
    _bedB = new BedFile(bedBFile);
    //_genome = new BedFile(genomeFile);
    _genome = new GenomeFile(genomeFile);


    TestOverlapsCUDA();
}


/*
Destructor
*/
BitsTestCUDA::~BitsTestCUDA(void) {
}



void BitsTestCUDA::TestOverlapsCUDA() {

	int *prt;
	hipMalloc(&prt, 0);

    vector<struct interval> A, B;
    read_and_map_files(_genome,
                       &_offsets,
                       _bedA,
                       _bedB,
                       &A,
                       &B);

	CHRPOS max_offset = 0;
	map<string,CHRPOS>::const_iterator itr;
	for (itr = _offsets.begin(); itr != _offsets.end(); ++itr)
		max_offset += itr->second;


	unsigned int O;
	double mean,sd,p;

    test_intersections_bsearch_cuda(&A[0],
								    A.size(),
								    &B[0],
								    B.size(),
								    _N,
								    max_offset,
								    &O,
								    &mean,
								    &sd,
								    &p);

	printf("O:%u\tE:%f\tsd:%f\tp:%f\n", O, mean, sd, p);
}
